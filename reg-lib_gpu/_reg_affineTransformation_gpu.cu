
#include <hip/hip_runtime.h>
/*
 *  _reg_affineTransformation_gpu.cu
 *
 *
 *  Created by Marc Modat on 25/03/2009.
 *  Copyright (c) 2009, University College London. All rights reserved.
 *  Centre for Medical Image Computing (CMIC)
 *  See the LICENSE.txt file in the nifty_reg root folder
 *
 */
#ifdef _USE_CUDA

#ifndef _REG_AFFINETRANSFORMATION_GPU_CU
#define _REG_AFFINETRANSFORMATION_GPU_CU

#include "_reg_affineTransformation_gpu.h"
#include "_reg_affineTransformation_gpu_kernels.cu"

/* *************************************************************** */
/* *************************************************************** */
void reg_affine_positionField_gpu(	mat44 *affineMatrix,
					nifti_image *targetImage,
					float4 **array_d)
{
	int3 imageSize = make_int3(targetImage->nx,targetImage->ny,targetImage->nz);
	CUDA_SAFE_CALL(cudaMemcpyToSymbol(c_ImageSize,&imageSize,sizeof(int3)));
	CUDA_SAFE_CALL(cudaMemcpyToSymbol(c_VoxelNumber,&(targetImage->nvox),sizeof(int)));

	// If the target sform is defined, it is used. The qform is used otherwise
	mat44 *targetMatrix;
	if(targetImage->sform_code>0)
		targetMatrix=&(targetImage->sto_xyz);
	else targetMatrix=&(targetImage->qto_xyz);

	// We here performed Affine * TargetMat * voxelIndex
	// Affine * TargetMat is constant
	mat44 transformationMatrix = reg_mat44_mul(affineMatrix, targetMatrix);

	// The transformation matrix is binded to a texture
	float4 *transformationMatrix_h;
    CUDA_SAFE_CALL(cudaMallocHost((void **)&transformationMatrix_h, 3*sizeof(float4)));
	float4 *transformationMatrix_d;
	CUDA_SAFE_CALL(cudaMalloc((void **)&transformationMatrix_d, 3*sizeof(float4)));
	for(int i=0; i<3; i++){
		transformationMatrix_h[i].x=transformationMatrix.m[i][0];
		transformationMatrix_h[i].y=transformationMatrix.m[i][1];
		transformationMatrix_h[i].z=transformationMatrix.m[i][2];
		transformationMatrix_h[i].w=transformationMatrix.m[i][3];
	}
	CUDA_SAFE_CALL(cudaMemcpy(transformationMatrix_d, transformationMatrix_h, 3*sizeof(float4), cudaMemcpyHostToDevice));
	cudaBindTexture(0,txAffineTransformation,transformationMatrix_d,3*sizeof(float4));
	
	const unsigned int Grid_reg_affine_deformationField = (unsigned int)ceil((float)targetImage->nvox/(float)Block_reg_affine_deformationField);
	dim3 B1(Block_reg_affine_deformationField,1,1);
	dim3 G1(Grid_reg_affine_deformationField,1,1);

	reg_affine_positionField_kernel <<< G1, B1 >>> (*array_d);
	CUDA_SAFE_CALL(cudaThreadSynchronize());
#if _VERBOSE
	printf("[VERBOSE] reg_affine_deformationField_kernel kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
	       cudaGetErrorString(cudaGetLastError()),G1.x,G1.y,G1.z,B1.x,B1.y,B1.z);
#endif
	
        // Unbind
	CUDA_SAFE_CALL(cudaUnbindTexture(txAffineTransformation));

	CUDA_SAFE_CALL(cudaFree(transformationMatrix_d));
	CUDA_SAFE_CALL(cudaFreeHost((void *)transformationMatrix_h));
}
/* *************************************************************** */
/* *************************************************************** */

#endif
#endif
